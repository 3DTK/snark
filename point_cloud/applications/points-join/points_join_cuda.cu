#include "hip/hip_runtime.h"
// This file is part of snark, a generic and flexible library for robotics research
// Copyright (c) 2011 The University of Sydney
// All rights reserved.
//
// Redistribution and use in source and binary forms, with or without
// modification, are permitted provided that the following conditions are met:
// 1. Redistributions of source code must retain the above copyright
//    notice, this list of conditions and the following disclaimer.
// 2. Redistributions in binary form must reproduce the above copyright
//    notice, this list of conditions and the following disclaimer in the
//    documentation and/or other materials provided with the distribution.
// 3. Neither the name of the University of Sydney nor the
//    names of its contributors may be used to endorse or promote products
//    derived from this software without specific prior written permission.
//
// NO EXPRESS OR IMPLIED LICENSES TO ANY PARTY'S PATENT RIGHTS ARE
// GRANTED BY THIS LICENSE.  THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT
// HOLDERS AND CONTRIBUTORS \"AS IS\" AND ANY EXPRESS OR IMPLIED
// WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE IMPLIED WARRANTIES OF
// MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE ARE
// DISCLAIMED. IN NO EVENT SHALL THE COPYRIGHT OWNER OR CONTRIBUTORS BE
// LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR
// CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF
// SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR
// BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY,
// WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT (INCLUDING NEGLIGENCE
// OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE OF THIS SOFTWARE, EVEN
// IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.

#include <assert.h>
#include <comma/base/exception.h>
#include <hip/hip_runtime.h>
#include "points_join_cuda.h"

__global__ void snark_cuda_squared_norms_impl( double x, double y, double z, const double *points, double *squared_norms, unsigned int size )
{
    unsigned int i = blockDim.x * blockIdx.x + threadIdx.x;
    if( i >= size ) { return; }
    unsigned int k = i * 3;
    x -= points[k];
    y -= points[k+1];
    z -= points[k+2];
    squared_norms[i] = x * x + y * y + z * z;
}

hipError_t snark_cuda_squared_norms( double x, double y, double z, const double *points, double *square_norms, unsigned int size )
{
    int threads = 128;
    int blocks = ( size - 1 ) / threads + 1;
    snark_cuda_squared_norms_impl<<<blocks, threads>>>( x, y, z, points, square_norms, size );
    return hipGetLastError();
}

namespace snark { namespace cuda {

void squared_norms( const Eigen::Vector3d& v, buffer& b )
{
    unsigned int size = b.out.size();
    hipError_t err = snark_cuda_squared_norms( v.x(), v.y(), v.z(), b.cuda_in, b.cuda_out, size ); // this call comprises 25% of (massive) overhead
    if( err != hipSuccess ) { COMMA_THROW( comma::exception, "cuda: square norm calculation failed; " << hipGetErrorString( err ) ); }
    err = hipMemcpy( &b.out[0], b.cuda_out, size * sizeof( double ), hipMemcpyDeviceToHost ); // this memcpy comprises 75% of (massive) overhead
    if( err != hipSuccess ) { COMMA_THROW( comma::exception, "cuda: copy failed; " << hipGetErrorString( err ) ); }
}

} } // namespace snark { namespace cuda {
